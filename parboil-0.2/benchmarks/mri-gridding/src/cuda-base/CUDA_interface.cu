#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include "parboil.h"

#include "UDTypes.h"
#include "scanLargeArray.h"
#include "GPU_kernels.cu"
#include "CPU_kernels.h"

#define USE_CUDPP 0
#if USE_CUDPP
#include "cudpp.h"
#else
#include "sort.h"
#include "scanLargeArray.h"
#endif

#define BLOCKSIZE 512
#define PI 3.14159265359
#define CUERR \
  do { \
    hipError_t err; \
    if ((err = hipGetLastError()) != hipSuccess) { \
      printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
      return; \
    } \
  } while (0)

/***********************************************************************
 * CUDA_interface is the main function for GPU execution. This
 * implementation uses compact binning to distribute input elements
 * into unit-cubed sized bins. The bins are then visited by GPU
 * threads, where every thread computes the value of one (or small set)
 * of output elements by computing the contributions of elements in 
 * neighboring bins to these output elements.
 *
 * The bins have a limited bin size and everything beyond that bin size
 * is offloaded to the CPU to be computed in parallel with the GPU
 * gridding.
 ***********************************************************************/
void CUDA_interface (
  struct pb_TimerSet* timers,
  unsigned int n,       // Number of input elements
  parameters params,    // Parameter struct which defines output gridSize, cutoff distance, etc.
  ReconstructionSample* sample, // Array of input elements
  float* LUT,           // Precomputed LUT table of Kaiser-Bessel function. 
                          // Used for computation on CPU instead of using the function every time
  int sizeLUT,          // Size of LUT
  cmplx* gridData,      // Array of output grid points. Each element has a real and imaginary component
  float* sampleDensity  // Array of same size as gridData couting the number of contributions
                          // to each grid point in the gridData array
){

  /* Initializing all variables */
  dim3 dims (8,4,2); //size of a gridding block on the GPU

  /* x, y, z dimensions of the output grid (gridData) */
  int size_x = params.gridSize[0];
  int size_y = params.gridSize[1];
  int size_z = params.gridSize[2];
  int size_xy = size_y*size_x;

  int gridNumElems = size_x * size_y * size_z;  // Total number of grid points

  float beta = PI * sqrt(4*params.kernelWidth*params.kernelWidth/(params.oversample*params.oversample) * (params.oversample-.5)*(params.oversample-.5)-.8);

  float cutoff = float(params.kernelWidth)/2.0; // cutoff radius
  float cutoff2 = cutoff*cutoff;                // square of cutoff radius
  float _1overCutoff2 = 1/cutoff2;              // 1 over square of cutoff radius

  /* Declarations of device data structures */
  ReconstructionSample* sample_d = NULL;    // Device array for original input array
  ReconstructionSample* sortedSample_d = NULL;             // Device array of the sorted (into bins) input elements.
                                            // This array is accessed by sortedSampleSoA_d in a structure
                                            //   of arrays manner.
  float2* gridData_d = NULL;                // Device array for output grid
  float* sampleDensity_d = NULL;            // Device array for output sample density
  unsigned int* idxKey_d = NULL;            // Array of bin indeces generated in the binning kernel
                                            //   and used to sort the input elements into their
                                            //   corresponding bins
  unsigned int* idxValue_d = NULL;          // This array holds the indices of input elements in the
                                            //   the original array. This array is sorted using the
                                            //   the idxKey_d array, and once sorted, it is used in
                                            //   the reorder kernel to move the actual elements into
                                            //   their corresponding bins.
  unsigned int* binCount_d = NULL;          // Zero-initialized array which counts the number of elements
                                            //   put in each bin. Based on this array, we determine which
                                            //   elements get offloaded to the CPU
  unsigned int* binStartAddr_d = NULL;      // Array of start offset of each of the compact bins

  /* Allocating device memory */
  pb_SwitchToTimer(timers, pb_TimerID_COPY);

  hipMalloc((void**)&sortedSample_d, n*sizeof(ReconstructionSample));
  hipMalloc((void**)&binStartAddr_d, (gridNumElems+1)*sizeof(unsigned int));
  hipMalloc((void**)&sample_d, n*sizeof(ReconstructionSample));
  hipMalloc((void**)&idxKey_d, (((n+3)/4)*4)*sizeof(unsigned int));   //Pad to nearest multiple of 4 to 
  hipMalloc((void**)&idxValue_d, (((n+3)/4)*4)*sizeof(unsigned int)); //satisfy a property of the sorting kernel.

/*The CUDPP library features highly optimizes implementations for radix sort
  and prefix sum. However for portability reasons, we implemented our own,
  slightly less optimized versions of these operations. When performing
  prefix sum using CUDPP, the output array has to be different from the input
  array, which is why we would allocate an array for binCount_d. For our
  implementation, we allow the input and output arrays to be the same,
  therefore we reuse the binCount_d array to get the starting offset of each
  bin. */
#if USE_CUDPP
  hipMalloc((void**)&binCount_d, (gridNumElems+1)*sizeof(unsigned int));
#else
  binCount_d = binStartAddr_d;
#endif
  CUERR;

  /* Transfering data from Host to Device */
  hipMemcpyToSymbol(HIP_SYMBOL(cutoff2_c), &cutoff2, sizeof(float), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(cutoff_c), &cutoff, sizeof(float), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(gridSize_c), params.gridSize, 3*sizeof(int), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(size_xy_c), &size_xy, sizeof(int), 0);
  hipMemcpyToSymbol(HIP_SYMBOL(_1overCutoff2_c), &_1overCutoff2, sizeof(float), 0);
  hipMemcpy(sample_d, sample, n*sizeof(ReconstructionSample), hipMemcpyHostToDevice);
  hipMemset(binCount_d, 0, (gridNumElems+1)*sizeof(unsigned int));

  // Initialize padding to max integer value, so that when sorted,
  // these elements get pushed to the end of the array.
  hipMemset(idxKey_d+n, 0xFF, (((n+3)&~(3))-n)*sizeof(unsigned int));

  pb_SwitchToTimer(timers, pb_TimerID_KERNEL);

  /* STEP 1: Perform binning. This kernel determines which output bin each input element
   * goes into. Any excess (beyond binsize) is put in the CPU bin
   */
  dim3 block1 (BLOCKSIZE);
  dim3 grid1 ((n+BLOCKSIZE-1)/BLOCKSIZE);

  binning_kernel<<<grid1, block1>>>(n, sample_d, idxKey_d, idxValue_d, binCount_d, params.binsize, gridNumElems);

  /* STEP 2: Sort the index-value pair generate in the binning kernel */
#if USE_CUDPP
  CUDPPConfiguration config;
  config.datatype = CUDPP_UINT;
  config.algorithm = CUDPP_SORT_RADIX;
  config.options = CUDPP_OPTION_KEY_VALUE_PAIRS;

  CUDPPHandle sortplan = 0;
  CUDPPResult result = cudppPlan(&sortplan, config, n, 1, 0);

  int precision = 0;
  int numElems = gridNumElems;
  while (numElems > 0){
    numElems >>= 1;
    precision++;
  }

  cudppSort(sortplan, idxKey_d, idxValue_d, int(precision), n);
  result = cudppDestroyPlan(sortplan);
#else
  sort(n, gridNumElems+1, idxKey_d, idxValue_d);
#endif

  /* STEP 3: Reorder the input data, based on the sorted values from Step 2.
   * this step also involves changing the data from array of structs to a struct
   * of arrays. Also in this kernel, we populate an array with the starting index
   * of every output bin features in the input array, based on the sorted indices 
   * from Step 2.
   * At the end of this step, we copy the start address and list of input elements
   * that will be computed on the CPU.
   */
  reorder_kernel<<<grid1,block1>>>(n, idxValue_d, sample_d, sortedSample_d);

  pb_SwitchToTimer(timers, pb_TimerID_COPY);

  hipFree(idxValue_d);
  hipFree(idxKey_d);
  hipFree(sample_d);

  pb_SwitchToTimer(timers, pb_TimerID_KERNEL);

  /* STEP 4: In this step we generate the ADD scan of the array of starting indices
   * of the output bins. The result is an array that contains the starting address of
   * every output bin.
   */
#if USE_CUDPP
  config.datatype = CUDPP_UINT;
  config.algorithm = CUDPP_SCAN;
  config.options = CUDPP_OPTION_EXCLUSIVE;
  config.op=CUDPP_ADD;

  CUDPPHandle scanplan = 0;
  result = cudppPlan(&scanplan, config, gridNumElems+1, 1, 0);

  cudppScan(scanplan, binCount_d, binStartAddr_d, gridNumElems+1);
  result = cudppDestroyPlan(scanplan);
#else
  scanLargeArray(gridNumElems+1, binCount_d);
#endif

  pb_SwitchToTimer(timers, pb_TimerID_COPY);

  // Copy back to the CPU the indices of the input elements that will be processed on the CPU
  int cpuStart;
  hipMemcpy(&cpuStart, binCount_d+gridNumElems, sizeof(unsigned int), hipMemcpyDeviceToHost);

  int CPUbin_size = int(n)-int(cpuStart);

  ReconstructionSample* CPUbin;
  hipHostMalloc((void**)&CPUbin,CPUbin_size*sizeof(ReconstructionSample));
  hipMemcpy(CPUbin, sortedSample_d+cpuStart, CPUbin_size*sizeof(ReconstructionSample), hipMemcpyDeviceToHost);

#if USE_CUDPP
  hipFree(binCount_d);
#endif

  /* STEP 5: Perform the binning on the GPU. The results are computed in a gather fashion
   * where each thread computes the value of one output element by reading the relevant
   * bins.
   */
  hipMalloc((void**)&gridData_d, gridNumElems*sizeof(float2));
  hipMalloc((void**)&sampleDensity_d, gridNumElems*sizeof(float));
  CUERR;

  hipMemset(gridData_d, 0, gridNumElems*sizeof(float2));
  hipMemset(sampleDensity_d, 0, gridNumElems*sizeof(float));

  pb_SwitchToTimer(timers, pb_TimerID_KERNEL);

  dim3 block2 (dims.x,dims.y,dims.z);
  dim3 grid2 (size_x/dims.x, (size_y*size_z)/(dims.y*dims.z));

  gridding_GPU<<<grid2, block2>>>(sortedSample_d, binStartAddr_d, gridData_d, sampleDensity_d, beta);

  pb_SwitchToTimer(timers, pb_TimerID_COPY);

  /* Copying the results from the Device to the Host */
  hipMemcpy(sampleDensity, sampleDensity_d, gridNumElems*sizeof(float),hipMemcpyDeviceToHost);
  hipMemcpy(gridData, gridData_d, gridNumElems*sizeof(float2),hipMemcpyDeviceToHost);

  pb_SwitchToTimer(timers, pb_TimerID_COMPUTE);

  /* STEP 6: Computing the contributions of the sample points handled by the Host
   * and adding those to the GPU results.
   */
  gridding_Gold(CPUbin_size, params, CPUbin, LUT, sizeLUT, gridData, sampleDensity);

  pb_SwitchToTimer(timers, pb_TimerID_COPY);

  hipHostFree(CPUbin);
  hipFree(gridData_d);
  hipFree(sampleDensity_d);
  hipFree(binCount_d);
  hipFree(sortedSample_d);

  pb_SwitchToTimer(timers, pb_TimerID_NONE);

  return;
}
