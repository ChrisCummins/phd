#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

#include "scanLargeArray.h"

#define UINT32_MAX 4294967295
#define BITS 4
#define LNB 4

#define SORT_BS 256

#define CONFLICT_FREE_OFFSET(index) ((index) >> LNB + (index) >> (2*LNB))
#define BLOCK_P_OFFSET (4*SORT_BS+1+(4*SORT_BS+1)/16+(4*SORT_BS+1)/64)

__device__ void scan (unsigned int s_data[BLOCK_P_OFFSET]){
  unsigned int thid = threadIdx.x;

  __syncthreads();

  s_data[2*thid+1+CONFLICT_FREE_OFFSET(2*thid+1)] += s_data[2*thid+CONFLICT_FREE_OFFSET(2*thid)];
  s_data[2*(blockDim.x+thid)+1+CONFLICT_FREE_OFFSET(2*(blockDim.x+thid)+1)] += s_data[2*(blockDim.x+thid)+CONFLICT_FREE_OFFSET(2*(blockDim.x+thid))];

  unsigned int stride = 2;
  for (unsigned int d = blockDim.x; d > 0; d >>= 1)
  {
    __syncthreads();

    if (thid < d)
    {
      unsigned int i  = 2*stride*thid;
      unsigned int ai = i + stride - 1;
      unsigned int bi = ai + stride;

      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      s_data[bi] += s_data[ai];
    }

    stride *= 2;
  }

  if (thid == 0){
    unsigned int last = 4*blockDim.x-1;
    last += CONFLICT_FREE_OFFSET(last);
    s_data[4*blockDim.x+CONFLICT_FREE_OFFSET(4*blockDim.x)] = s_data[last];
    s_data[last] = 0;
  }

  for (unsigned int d = 1; d <= blockDim.x; d *= 2)
  {
    stride >>= 1;

    __syncthreads();

    if (thid < d)
    {
      unsigned int i  = 2*stride*thid;
      unsigned int ai = i + stride - 1;
      unsigned int bi = ai + stride;

      ai += CONFLICT_FREE_OFFSET(ai);
      bi += CONFLICT_FREE_OFFSET(bi);

      unsigned int t  = s_data[ai];
      s_data[ai] = s_data[bi];
      s_data[bi] += t;
    }
  }
  __syncthreads();

  unsigned int temp = s_data[2*thid+CONFLICT_FREE_OFFSET(2*thid)];
  s_data[2*thid+CONFLICT_FREE_OFFSET(2*thid)] = s_data[2*thid+1+CONFLICT_FREE_OFFSET(2*thid+1)];
  s_data[2*thid+1+CONFLICT_FREE_OFFSET(2*thid+1)] += temp;

  unsigned int temp2 = s_data[2*(blockDim.x+thid)+CONFLICT_FREE_OFFSET(2*(blockDim.x+thid))];
  s_data[2*(blockDim.x+thid)+CONFLICT_FREE_OFFSET(2*(blockDim.x+thid))] = s_data[2*(blockDim.x+thid)+1+CONFLICT_FREE_OFFSET(2*(blockDim.x+thid)+1)];
  s_data[2*(blockDim.x+thid)+1+CONFLICT_FREE_OFFSET(2*(blockDim.x+thid)+1)] += temp2;

  __syncthreads();
}

__global__ static void splitSort(int numElems, int iter, unsigned int* keys, unsigned int* values, unsigned int* histo)
{
    __shared__ unsigned int flags[BLOCK_P_OFFSET];
    __shared__ unsigned int histo_s[1<<BITS];

    const unsigned int tid = threadIdx.x;
    const unsigned int gid = blockIdx.x*4*SORT_BS+4*threadIdx.x;

    // Copy input to shared mem. Assumes input is always even numbered
    uint4 lkey = { UINT32_MAX, UINT32_MAX, UINT32_MAX, UINT32_MAX};
    uint4 lvalue;
    if (gid < numElems){
      lkey = *((uint4*)(keys+gid));
      lvalue = *((uint4*)(values+gid));
    }

    if(tid < (1<<BITS)){
      histo_s[tid] = 0;
    }
    __syncthreads();

    atomicAdd(histo_s+((lkey.x&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)),1);
    atomicAdd(histo_s+((lkey.y&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)),1);
    atomicAdd(histo_s+((lkey.z&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)),1);
    atomicAdd(histo_s+((lkey.w&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)),1);

    uint4 index = {4*tid, 4*tid+1, 4*tid+2, 4*tid+3};

    for (int i=BITS*iter; i<BITS*(iter+1);i++){
      const uint4 flag = {(lkey.x>>i)&0x1,(lkey.y>>i)&0x1,(lkey.z>>i)&0x1,(lkey.w>>i)&0x1};

      flags[index.x+CONFLICT_FREE_OFFSET(index.x)] = 1<<(16*flag.x);
      flags[index.y+CONFLICT_FREE_OFFSET(index.y)] = 1<<(16*flag.y);
      flags[index.z+CONFLICT_FREE_OFFSET(index.z)] = 1<<(16*flag.z);
      flags[index.w+CONFLICT_FREE_OFFSET(index.w)] = 1<<(16*flag.w);

      scan (flags);

      index.x = (flags[index.x+CONFLICT_FREE_OFFSET(index.x)]>>(16*flag.x))&0xFFFF;
      index.y = (flags[index.y+CONFLICT_FREE_OFFSET(index.y)]>>(16*flag.y))&0xFFFF;
      index.z = (flags[index.z+CONFLICT_FREE_OFFSET(index.z)]>>(16*flag.z))&0xFFFF;
      index.w = (flags[index.w+CONFLICT_FREE_OFFSET(index.w)]>>(16*flag.w))&0xFFFF;

      unsigned short offset = flags[4*blockDim.x+CONFLICT_FREE_OFFSET(4*blockDim.x)]&0xFFFF;
      index.x += (flag.x) ? offset : 0;
      index.y += (flag.y) ? offset : 0;
      index.z += (flag.z) ? offset : 0;
      index.w += (flag.w) ? offset : 0;

      __syncthreads();
    }

    // Write result.
    if (gid < numElems){
      keys[blockIdx.x*4*SORT_BS+index.x] = lkey.x;
      keys[blockIdx.x*4*SORT_BS+index.y] = lkey.y;
      keys[blockIdx.x*4*SORT_BS+index.z] = lkey.z;
      keys[blockIdx.x*4*SORT_BS+index.w] = lkey.w;

      values[blockIdx.x*4*SORT_BS+index.x] = lvalue.x;
      values[blockIdx.x*4*SORT_BS+index.y] = lvalue.y;
      values[blockIdx.x*4*SORT_BS+index.z] = lvalue.z;
      values[blockIdx.x*4*SORT_BS+index.w] = lvalue.w;
    }
    if (tid < (1<<BITS)){
      histo[gridDim.x*threadIdx.x+blockIdx.x] = histo_s[tid];
    }
}

__global__ void splitRearrange (int numElems, int iter, unsigned int* keys_i, unsigned int* keys_o, unsigned int* values_i, unsigned int* values_o, unsigned int* histo){
  __shared__ unsigned int histo_s[(1<<BITS)];
  __shared__ unsigned int array_s[4*SORT_BS];
  int index = blockIdx.x*4*SORT_BS + 4*threadIdx.x;

  if (threadIdx.x < (1<<BITS)){
    histo_s[threadIdx.x] = histo[gridDim.x*threadIdx.x+blockIdx.x];
  }

  uint4 mine, value;
  if (index < numElems){
    mine = *((uint4*)(keys_i+index));
    value = *((uint4*)(values_i+index));
  } else {
    mine.x = UINT32_MAX;
    mine.y = UINT32_MAX;
    mine.z = UINT32_MAX;
    mine.w = UINT32_MAX;
  }
  uint4 masks = {(mine.x&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
                 (mine.y&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
                 (mine.z&((1<<(BITS*(iter+1)))-1))>>(BITS*iter),
                 (mine.w&((1<<(BITS*(iter+1)))-1))>>(BITS*iter)};

  ((uint4*)array_s)[threadIdx.x] = masks;
  __syncthreads();

  uint4 new_index = {histo_s[masks.x],histo_s[masks.y],histo_s[masks.z],histo_s[masks.w]};

  int i = 4*threadIdx.x-1;
  while (i >= 0){
    if (array_s[i] == masks.x){
      new_index.x++;
      i--;
    } else {
      break;
    }
  }

  new_index.y = (masks.y == masks.x) ? new_index.x+1 : new_index.y;
  new_index.z = (masks.z == masks.y) ? new_index.y+1 : new_index.z;
  new_index.w = (masks.w == masks.z) ? new_index.z+1 : new_index.w;

  if (index < numElems){
    keys_o[new_index.x] = mine.x;
    values_o[new_index.x] = value.x;

    keys_o[new_index.y] = mine.y;
    values_o[new_index.y] = value.y;

    keys_o[new_index.z] = mine.z;
    values_o[new_index.z] = value.z;

    keys_o[new_index.w] = mine.w;
    values_o[new_index.w] = value.w;
  }
}

void sort (int numElems, unsigned int max_value, unsigned int* &dkeys, unsigned int* &dvalues){
  dim3 grid ((numElems+4*SORT_BS-1)/(4*SORT_BS));
  dim3 block (SORT_BS);

  unsigned int iterations = 0;
  while(max_value > 0){
    max_value >>= BITS;
    iterations++;
  }

  unsigned int *dhisto;
  unsigned int *dkeys_o, *dvalues_o;

  hipMalloc((void**)&dhisto, (1<<BITS)*grid.x*sizeof(unsigned int));
  hipMalloc((void**)&dkeys_o, numElems*sizeof(unsigned int));
  hipMalloc((void**)&dvalues_o, numElems*sizeof(unsigned int));

  for (int i=0; i<iterations; i++){
    splitSort<<<grid,block>>>(numElems, i, dkeys, dvalues, dhisto);

    scanLargeArray(grid.x*(1<<BITS), dhisto);

    splitRearrange<<<grid,block>>>(numElems, i, dkeys, dkeys_o, dvalues, dvalues_o, dhisto);

    unsigned int* temp = dkeys;
    dkeys = dkeys_o;
    dkeys_o = temp;

    temp = dvalues;
    dvalues = dvalues_o;
    dvalues_o = temp;
  }

  hipFree(dkeys_o);
  hipFree(dvalues_o);
  hipFree(dhisto);
}
