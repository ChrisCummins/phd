/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define BLOCK_SIZE 1024
#define GRID_SIZE 65535
#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2*LOG_NUM_BANKS))
#define EXPANDED_SIZE(__x) (__x+(__x>>LOG_NUM_BANKS)+(__x>>(2*LOG_NUM_BANKS)))

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void scan_L1_kernel(unsigned int n, unsigned int* data, unsigned int* inter)
{
    __shared__ unsigned int s_data[EXPANDED_SIZE(BLOCK_SIZE)]; 

    unsigned int thid = threadIdx.x;
    unsigned int g_ai = blockIdx.x*2*blockDim.x + threadIdx.x;
    unsigned int g_bi = g_ai + blockDim.x;

    unsigned int s_ai = thid;
    unsigned int s_bi = thid + blockDim.x;

    s_ai += CONFLICT_FREE_OFFSET(s_ai);
    s_bi += CONFLICT_FREE_OFFSET(s_bi);

    s_data[s_ai] = (g_ai < n) ? data[g_ai] : 0;
    s_data[s_bi] = (g_bi < n) ? data[g_bi] : 0;

    unsigned int stride = 1;
    for (unsigned int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)
        {
            unsigned int i  = 2*stride*thid;
            unsigned int ai = i + stride - 1;
            unsigned int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    if (thid == 0){
        unsigned int last = blockDim.x*2 -1;
        last += CONFLICT_FREE_OFFSET(last);
        inter[blockIdx.x] = s_data[last];
        s_data[last] = 0;
    }

    for (unsigned int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            unsigned int i  = 2*stride*thid;
            unsigned int ai = i + stride - 1;
            unsigned int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            unsigned int t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
    __syncthreads();

    if (g_ai < n) { data[g_ai] = s_data[s_ai]; }
    if (g_bi < n) { data[g_bi] = s_data[s_bi]; }
}

__global__ void scan_inter1_kernel(unsigned int* data, unsigned int iter)
{
    extern __shared__ unsigned int s_data[];

    unsigned int thid = threadIdx.x;
    unsigned int gthid = (blockIdx.x*blockDim.x + threadIdx.x);
    unsigned int gi = 2*iter*gthid;
    unsigned int g_ai = gi + iter - 1;
    unsigned int g_bi = g_ai + iter;

    unsigned int s_ai = 2*thid;
    unsigned int s_bi = 2*thid + 1;

    s_ai += CONFLICT_FREE_OFFSET(s_ai);
    s_bi += CONFLICT_FREE_OFFSET(s_bi);

    s_data[s_ai] = data[g_ai];
    s_data[s_bi] = data[g_bi];

    unsigned int stride = 1;
    for (unsigned int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)
        {
            unsigned int i  = 2*stride*thid;
            unsigned int ai = i + stride - 1;
            unsigned int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);
            s_data[bi] += s_data[ai];
        }

        stride *= 2;
    }

    __syncthreads();

    data[g_ai] = s_data[s_ai];
    data[g_bi] = s_data[s_bi];
}

__global__ void scan_inter2_kernel(unsigned int* data, unsigned int iter)
{
    extern __shared__ unsigned int s_data[];

    unsigned int thid = threadIdx.x;
    unsigned int gthid = (blockIdx.x*blockDim.x + threadIdx.x);
    unsigned int gi = 2*iter*gthid;
    unsigned int g_ai = gi + iter - 1;
    unsigned int g_bi = g_ai + iter;

    unsigned int s_ai = 2*thid;
    unsigned int s_bi = 2*thid + 1;

    s_ai += CONFLICT_FREE_OFFSET(s_ai);
    s_bi += CONFLICT_FREE_OFFSET(s_bi);

    s_data[s_ai] = data[g_ai];
    s_data[s_bi] = data[g_bi];

    unsigned int stride = blockDim.x*2;

    for (unsigned int d = 1; d <= blockDim.x; d *= 2)
    {
        stride >>= 1;

        __syncthreads();

        if (thid < d)
        {
            unsigned int i  = 2*stride*thid;
            unsigned int ai = i + stride - 1;
            unsigned int bi = ai + stride;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            unsigned int t  = s_data[ai];
            s_data[ai] = s_data[bi];
            s_data[bi] += t;
        }
    }
    __syncthreads();

    data[g_ai] = s_data[s_ai];
    data[g_bi] = s_data[s_bi];
}

__global__ void uniformAdd(unsigned int n, unsigned int *data, unsigned int *inter)
{

    __shared__ unsigned int uni;
    if (threadIdx.x == 0) { uni = inter[blockIdx.x]; }
    __syncthreads();

    unsigned int g_ai = blockIdx.x*2*blockDim.x + threadIdx.x;
    unsigned int g_bi = g_ai + blockDim.x;

    if (g_ai < n) { data[g_ai] += uni; }
    if (g_bi < n) { data[g_bi] += uni; }
}

void scanLargeArray( unsigned int gridNumElements, unsigned int* data_d) {
    unsigned int gridNumElems = gridNumElements;    

    // allocate device memory input and output arrays
    unsigned int* inter_d = NULL;

    // Run the prescan
    unsigned int size = (gridNumElems+BLOCK_SIZE-1)/BLOCK_SIZE;

    unsigned int dim_block;
    unsigned int current_max = size*BLOCK_SIZE;
    for (int block_size = 128; block_size <= BLOCK_SIZE; block_size *= 2){
      unsigned int array_size = block_size;
      while(array_size < size){
        array_size *= block_size;
      }
      if (array_size <= current_max){
        current_max = array_size;
        dim_block = block_size;
      }
    }

    hipMalloc( (void**) &inter_d, current_max*sizeof(unsigned int));
    hipMemset (inter_d, 0, current_max*sizeof(unsigned int));

    for (unsigned int i=0; i < (size+GRID_SIZE-1)/GRID_SIZE; i++){
        unsigned int gridSize = ((size-(i*GRID_SIZE)) > GRID_SIZE) ? GRID_SIZE : (size-i*GRID_SIZE);
        unsigned int numElems = ((gridNumElems-(i*GRID_SIZE*BLOCK_SIZE)) > (GRID_SIZE*BLOCK_SIZE)) ? (GRID_SIZE*BLOCK_SIZE) : (gridNumElems-(i*GRID_SIZE*BLOCK_SIZE));

        dim3 block (BLOCK_SIZE/2);
        dim3 grid (gridSize);
        scan_L1_kernel<<<grid, block>>>(numElems, data_d+(i*GRID_SIZE*BLOCK_SIZE), inter_d+(i*GRID_SIZE));
    }

    unsigned int stride = 1;
    for (unsigned int d = current_max; d > 1; d /= dim_block)
    {
        dim3 block (dim_block/2);
        dim3 grid (d/dim_block);

        scan_inter1_kernel<<<grid, block, EXPANDED_SIZE(dim_block)*sizeof(unsigned int)>>>(inter_d, stride);

        stride *= dim_block;
    }

    hipMemset(&(inter_d[current_max-1]), 0, sizeof(unsigned int));

    for (unsigned int d = dim_block; d <= current_max; d *= dim_block)
    {
        stride /= dim_block;
        dim3 block (dim_block/2);
        dim3 grid (d/dim_block);

        scan_inter2_kernel<<<grid, block, EXPANDED_SIZE(dim_block)*sizeof(unsigned int)>>>(inter_d, stride);
    }

    for (unsigned int i=0; i < (size+GRID_SIZE-1)/GRID_SIZE; i++){
        unsigned int gridSize = ((size-(i*GRID_SIZE)) > GRID_SIZE) ? GRID_SIZE : (size-i*GRID_SIZE);
        unsigned int numElems = ((gridNumElems-(i*GRID_SIZE*BLOCK_SIZE)) > (GRID_SIZE*BLOCK_SIZE)) ? (GRID_SIZE*BLOCK_SIZE) : (gridNumElems-(i*GRID_SIZE*BLOCK_SIZE));

        dim3 block (BLOCK_SIZE/2);
        dim3 grid (gridSize);

        uniformAdd<<<grid, block>>>(numElems, data_d+(i*GRID_SIZE*BLOCK_SIZE), inter_d+(i*GRID_SIZE));
    }

    hipFree(inter_d);
}
