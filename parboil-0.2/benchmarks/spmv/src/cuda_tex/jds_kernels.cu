#include "hip/hip_runtime.h"
#define WARP_BITS 5
#define WARP_SIZE 32
__global__ void spmv_jds_texture(float *dst_vector,
							   const float *d_data,const int *d_index, const int *d_perm,
							   const float *x_vec,const int *d_nzcnt,const int dim)
{
	int ix=blockIdx.x*blockDim.x+threadIdx.x;
	int warp_id= ix>>WARP_BITS;

	if(ix<dim)
	{
		float sum=0.0f;	
		int pt =d_perm[ix];
		int	bound=sh_zcnt_int[warp_id];
		//prefetch 0
		int j=jds_ptr_int[0]+ix;   
		float d = d_data[j]; 
		int i = d_index[j]; 
		float t =tex1Dfetch(tex_x_float,i);  
		
		if (bound>1)  //bound >=2
		{
			//prefetch 1
			j=jds_ptr_int[1]+ix;    
			i =  d_index[j];  
			int in;
			float dn;
			float tn;
			
			for(int k=2;k<bound;k++ )
			{	
				//prefetch k-1
				dn = d_data[j]; 
				//prefetch k
				j=jds_ptr_int[k]+ix;     
				in = d_index[j]; 
				//prefetch k-1
				tn =tex1Dfetch(tex_x_float,i);  
				
				//compute k-2 data
				sum += d*t;    
				//sweep to k
				i = in; 
				//sweep to k-1
				d = dn;  
				t =tn;   
			}	
			
			//fetch last
			dn = d_data[j];   
			//fetch last 
			tn =tex1Dfetch(tex_x_float,i); 
			//compute last -1
			sum += d*t;  
			//sweep to last
			d=dn;
			t=tn;
		}
		//compute last one
		sum += d*t; 
		
		//write out data 
		dst_vector[pt]=sum; 
		
	}
}


