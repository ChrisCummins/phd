
/***************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/
#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>

#include "file.h"
#include "common.h"
#include "cuerr.h"
#include "kernels.cu"

static int read_data(float *A0, int nx,int ny,int nz,FILE *fp) 
{	
	int s=0;
	for(int i=0;i<nz;i++)
	{
		for(int j=0;j<ny;j++)
		{
			for(int k=0;k<nx;k++)
			{
                                fread(A0+s,sizeof(float),1,fp);
				s++;
			}
		}
	}
	return 0;
}

int main(int argc, char** argv) {
	struct pb_TimerSet timers;
	struct pb_Parameters *parameters;
	

	
	printf("CUDA accelerated 7 points stencil codes****\n");
	printf("Original version by Li-Wen Chang <lchang20@illinois.edu> and I-Jui Sung<sung10@illinois.edu>\n");
	printf("This version maintained by Chris Rodrigues  ***********\n");
	parameters = pb_ReadParameters(&argc, argv);

	pb_InitializeTimerSet(&timers);
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	//declaration
	int nx,ny,nz;
	int size;
    int iteration;
	float c0=1.0f/6.0f;
	float c1=1.0f/6.0f/6.0f;

	if (argc<5) 
    {
      printf("Usage: probe nx ny nz tx ty t\n"
	     "nx: the grid size x\n"
	     "ny: the grid size y\n"
	     "nz: the grid size z\n"
		  "t: the iteration time\n");
      return -1;
    }

	nx = atoi(argv[1]);
	if (nx<1)
		return -1;
	ny = atoi(argv[2]);
	if (ny<1)
		return -1;
	nz = atoi(argv[3]);
	if (nz<1)
		return -1;
	iteration = atoi(argv[4]);
	if(iteration<1)
		return -1;

	
	//host data
	float *h_A0;
	float *h_Anext;
	//device
	float *d_A0;
	float *d_Anext;

	
	
	size=nx*ny*nz;
	
	h_A0=(float*)malloc(sizeof(float)*size);
	h_Anext=(float*)malloc(sizeof(float)*size);
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
        FILE *fp = fopen(parameters->inpFiles[0], "rb");
	read_data(h_A0, nx,ny,nz,fp);
        fclose(fp);
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//memory allocation
	hipMalloc((void **)&d_A0, size*sizeof(float));
	hipMalloc((void **)&d_Anext, size*sizeof(float));
	hipMemset(d_Anext,0,size*sizeof(float));

	//memory copy
	hipMemcpy(d_A0, h_A0, size*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_Anext, d_A0, size*sizeof(float), hipMemcpyDeviceToDevice);
	
	hipDeviceSynchronize();
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

	//only use tx-by-ty threads
	int tx=512;
	int ty=2;
	dim3 block (tx, ty, 1);
	dim3 grid ((nx+tx-1)/tx, (ny+ty-1)/ty,1);

	//main execution
	pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
	for(int t=0;t<iteration;t++)
	{
		block2D_reg_tiling<<<grid, block>>>(c0,c1, d_A0, d_Anext, nx, ny,  nz);
    float *d_temp=d_A0;
    d_A0 = d_Anext;
    d_Anext = d_temp;

	}
  CUERR // check and clear any existing errors
  float *d_temp=d_A0;
  d_A0 = d_Anext;
  d_Anext = d_temp;
	
	
	
	hipDeviceSynchronize();
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	hipMemcpy(h_Anext, d_Anext,size*sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipFree(d_A0);
    hipFree(d_Anext);
 
	if (parameters->outFile) {
		 pb_SwitchToTimer(&timers, pb_TimerID_IO);
		outputData(parameters->outFile,h_Anext,nx,ny,nz);
		
	}
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
		
	free (h_A0);
	free (h_Anext);
	pb_SwitchToTimer(&timers, pb_TimerID_NONE);

	pb_PrintTimerSet(&timers);
	pb_FreeParameters(parameters);

	return 0;

}
