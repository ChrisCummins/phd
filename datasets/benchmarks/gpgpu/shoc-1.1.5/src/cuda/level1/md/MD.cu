#include "hip/hip_runtime.h"
#include <cassert>
#include <cfloat>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <list>
#include <math.h>
#include <stdlib.h>
#include "cudacommon.h"
#include "MD.h"
#include "OptionParser.h"
#include "ResultDatabase.h"
#include "Utility.h"

using namespace std;

// Forward Declarations
template <class T, class forceVecType, class posVecType, bool useTexture,
          typename texReader>
void runTest(const string& testName, ResultDatabase& resultDB,
          OptionParser& op);

template <class T, class posVecType>
inline T distance(const posVecType* position, const int i, const int j);

template <class T>
inline void insertInOrder(std::list<T>& currDist, std::list<int>& currList,
        const int j, const T distIJ, const int maxNeighbors);

template <class T, class posVecType>
inline int buildNeighborList(const int nAtom, const posVecType* position,
        int* neighborList);

template <class T>
inline int populateNeighborList(std::list<T>& currDist,
        std::list<int>& currList, const int j, const int nAtom,
        int* neighborList);

// Texture caches for position info
texture<float4, 1, hipReadModeElementType> posTexture;
texture<int4, 1, hipReadModeElementType> posTexture_dp;

struct texReader_sp {
   __device__ __forceinline__ float4 operator()(int idx) const
   {
       return tex1Dfetch(posTexture, idx);
   }
};

// CUDA doesn't support double4 textures, so we have to do some conversion
// here, resulting in a bit of overhead, but it's still faster than
// an uncoalesced read
struct texReader_dp {
   __device__ __forceinline__ double4 operator()(int idx) const
   {
#if (__CUDA_ARCH__ < 130)
       // Devices before arch 130 don't support DP, and having the
       // __hiloint2double() intrinsic will cause compilation to fail.
       // This return statement added as a workaround -- it will compile,
       // but since the arch doesn't support DP, it will never be called
       return make_double4(0., 0., 0., 0.);
#else
       int4 v = tex1Dfetch(posTexture_dp, idx*2);
       double2 a = make_double2(__hiloint2double(v.y, v.x),
                                __hiloint2double(v.w, v.z));

       v = tex1Dfetch(posTexture_dp, idx*2 + 1);
       double2 b = make_double2(__hiloint2double(v.y, v.x),
                                __hiloint2double(v.w, v.z));

       return make_double4(a.x, a.y, b.x, b.y);
#endif
   }
};

// ****************************************************************************
// Function: compute_lj_force
//
// Purpose:
//   GPU kernel to calculate Lennard Jones force
//
// Arguments:
//      force3:     array to store the calculated forces
//      position:   positions of atoms
//      neighCount: number of neighbors for each atom to consider
//      neighList:  atom neighbor list
//      cutsq:      cutoff distance squared
//      lj1, lj2:   LJ force constants
//      inum:       total number of atoms
//
// Returns: nothing
//
// Programmer: Kyle Spafford
// Creation: July 26, 2010
//
// Modifications:
//
// ****************************************************************************
template <class T, class forceVecType, class posVecType, bool useTexture,
          typename texReader>
__global__ void compute_lj_force(forceVecType* __restrict__ force3,
                                 const posVecType* __restrict__ position,
                                 const int neighCount,
                                 const int* __restrict__ neighList,
                                 const T cutsq,
                                 const T lj1,
                                 const T lj2,
                                 const int inum)
{
    // Global ID - one thread per atom
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    // Position of this thread's atom
    posVecType ipos = position[idx];

    // Force accumulator
    forceVecType f = {0.0f, 0.0f, 0.0f};

    texReader positionTexReader;

    int j = 0;
    while (j < neighCount)
    {
        int jidx = neighList[j*inum + idx];
        posVecType jpos;
        if (useTexture)
        {
            // Use texture mem as a cache
            jpos = positionTexReader(jidx);
        }
        else
        {
            jpos = position[jidx];
        }

        // Calculate distance
        T delx = ipos.x - jpos.x;
        T dely = ipos.y - jpos.y;
        T delz = ipos.z - jpos.z;
        T r2inv = delx*delx + dely*dely + delz*delz;

        // If distance is less than cutoff, calculate force
        // and add to accumulator
        if (r2inv < cutsq)
        {
            r2inv = 1.0f/r2inv;
            T r6inv = r2inv * r2inv * r2inv;
            T force = r2inv*r6inv*(lj1*r6inv - lj2);

            f.x += delx * force;
            f.y += dely * force;
            f.z += delz * force;
        }
        j++;
    }

    // store the results
    force3[idx] = f;
}

// ****************************************************************************
// Function: checkResults
//
// Purpose:
//   Check device results against cpu results -- this is the CPU equivalent of
//
// Arguments:
//      d_force:   forces calculated on the device
//      position:  positions of atoms
//      neighList: atom neighbor list
//      nAtom:     number of atoms
// Returns:  true if results match, false otherwise
//
// Programmer: Kyle Spafford
// Creation: July 26, 2010
//
// Modifications:
//
// ****************************************************************************
template <class T, class forceVecType, class posVecType>
bool checkResults(forceVecType* d_force, posVecType *position,
                  int *neighList, int nAtom)
{
    for (int i = 0; i < nAtom; i++)
    {
        posVecType ipos = position[i];
        forceVecType f = {0.0f, 0.0f, 0.0f};
        int j = 0;
        while (j < maxNeighbors)
        {
            int jidx = neighList[j*nAtom + i];
            posVecType jpos = position[jidx];
            // Calculate distance
            T delx = ipos.x - jpos.x;
            T dely = ipos.y - jpos.y;
            T delz = ipos.z - jpos.z;
            T r2inv = delx*delx + dely*dely + delz*delz;

            // If distance is less than cutoff, calculate force
            if (r2inv < cutsq) {

                r2inv = 1.0f/r2inv;
                T r6inv = r2inv * r2inv * r2inv;
                T force = r2inv*r6inv*(lj1*r6inv - lj2);

                f.x += delx * force;
                f.y += dely * force;
                f.z += delz * force;
            }
            j++;
        }
        // Check the results
        T diffx = (d_force[i].x - f.x) / d_force[i].x;
        T diffy = (d_force[i].y - f.y) / d_force[i].y;
        T diffz = (d_force[i].z - f.z) / d_force[i].z;
        T err = sqrt(diffx*diffx) + sqrt(diffy*diffy) + sqrt(diffz*diffz);
        if (err > (3.0 * EPSILON))
        {
            cout << "Test Failed, idx: " << i << " diff: " << err << "\n";
            cout << "f.x: " << f.x << " df.x: " << d_force[i].x << "\n";
            cout << "f.y: " << f.y << " df.y: " << d_force[i].y << "\n";
            cout << "f.z: " << f.z << " df.z: " << d_force[i].z << "\n";
            cout << "Test FAILED\n";
            return false;
        }
    }
    cout << "Test Passed\n";
    return true;
}


// ********************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ********************************************************
void
addBenchmarkSpecOptions(OptionParser &op)
{
    op.addOption("iterations", OPT_INT, "1",
                     "specify MD kernel iterations", 'r');
}

// ********************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the md benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ********************************************************
void
RunBenchmark(ResultDatabase &resultDB, OptionParser &op)
{
    // Test to see if this device supports double precision
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    cout << "Running single precision test" << endl;
    runTest<float, float3, float4, true, texReader_sp>("MD-LJ", resultDB, op);
    if ((deviceProp.major == 1 && deviceProp.minor >= 3) ||
               (deviceProp.major >= 2))
    {
        cout << "Running double precision test" << endl;
        runTest<double, double3, double4, true, texReader_dp>
            ("MD-LJ-DP", resultDB, op);
    } else {
        cout << "Skipping double precision test" << endl;
        char atts[32] = "DP_Not_Supported";
        // resultDB requires neg entry for every possible result
        int passes = op.getOptionInt("passes");
        for (int i = 0; i < passes; i++) {
            resultDB.AddResult("MD-LJ-DP" , atts, "GB/s", FLT_MAX);
            resultDB.AddResult("MD-LJ-DP_PCIe" , atts, "GB/s", FLT_MAX);
            resultDB.AddResult("MD-LJ-DP-Bandwidth", atts, "GB/s", FLT_MAX);
            resultDB.AddResult("MD-LJ-DP-Bandwidth_PCIe", atts, "GB/s", FLT_MAX);
            resultDB.AddResult("MD-LJ-DP_Parity" , atts, "GB/s", FLT_MAX);
        }
    }
}

template <class T, class forceVecType, class posVecType, bool useTexture,
          typename texReader>
void runTest(const string& testName, ResultDatabase& resultDB, OptionParser& op)
{
    // Problem Parameters
    const int probSizes[4] = { 12288, 24576, 36864, 73728 };
    int sizeClass = op.getOptionInt("size");
    assert(sizeClass >= 0 && sizeClass < 5);
    int nAtom = probSizes[sizeClass - 1];

    // Allocate problem data on host
    posVecType*   position;
    forceVecType* force;
    int* neighborList;

    CUDA_SAFE_CALL(hipHostMalloc((void**)&position, nAtom*sizeof(posVecType)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&force,    nAtom*sizeof(forceVecType)));
    CUDA_SAFE_CALL(hipHostMalloc((void**)&neighborList,
            nAtom*maxNeighbors*sizeof(int)));

    // Allocate device memory for position and force
    forceVecType* d_force;
    posVecType*   d_position;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_force,    nAtom*sizeof(forceVecType)));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_position, nAtom*sizeof(posVecType)));

    // Allocate device memory for neighbor list
    int* d_neighborList;
    CUDA_SAFE_CALL(hipMalloc((void**)&d_neighborList,
                              nAtom*maxNeighbors*sizeof(int)));

    cout << "Initializing test problem (this can take several "
            "minutes for large problems)\n";

    // Seed random number generator
    srand48(8650341L);

    // Initialize positions -- random distribution in cubic domain
    // domainEdge constant specifies edge length
    for (int i = 0; i < nAtom; i++)
    {
        position[i].x = (T)(drand48() * domainEdge);
        position[i].y = (T)(drand48() * domainEdge);
        position[i].z = (T)(drand48() * domainEdge);
    }

    if (useTexture)
    {
        // Set up 1D texture to cache position info
        hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();

        // Bind a 1D texture to the position array
        CUDA_SAFE_CALL(hipBindTexture(0, posTexture, d_position, channelDesc,
                nAtom*sizeof(float4)));

        hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<int4>();

        // Bind a 1D texture to the position array
        CUDA_SAFE_CALL(hipBindTexture(0, posTexture_dp, d_position,
                channelDesc2, nAtom*sizeof(double4)));
    }

    // Keep track of how many atoms are within the cutoff distance to
    // accurately calculate FLOPS later
    int totalPairs = buildNeighborList<T, posVecType>(nAtom, position,
            neighborList);

    cout << "Finished.\n";
    cout << totalPairs << " of " << nAtom*maxNeighbors <<
            " pairs within cutoff distance = " <<
            100.0 * ((double)totalPairs / (nAtom*maxNeighbors)) << " %" << endl;

    // Time the transfer of input data to the GPU
    hipEvent_t inputTransfer_start, inputTransfer_stop;
    hipEventCreate(&inputTransfer_start);
    hipEventCreate(&inputTransfer_stop);

    hipEventRecord(inputTransfer_start, 0);
    // Copy neighbor list data to GPU
    CUDA_SAFE_CALL(hipMemcpy(d_neighborList, neighborList,
            maxNeighbors*nAtom*sizeof(int), hipMemcpyHostToDevice));
    // Copy position to GPU
    CUDA_SAFE_CALL(hipMemcpy(d_position, position, nAtom*sizeof(posVecType),
            hipMemcpyHostToDevice));
    hipEventRecord(inputTransfer_stop, 0);
    CUDA_SAFE_CALL(hipEventSynchronize(inputTransfer_stop));

    // Get elapsed time
    float inputTransfer_time = 0.0f;
    hipEventElapsedTime(&inputTransfer_time, inputTransfer_start,
            inputTransfer_stop);
    inputTransfer_time *= 1.e-3;

    int blockSize = 256;
    int gridSize  = nAtom / blockSize;

    // Warm up the kernel and check correctness
    compute_lj_force<T, forceVecType, posVecType, useTexture, texReader>
                    <<<gridSize, blockSize>>>
                    (d_force, d_position, maxNeighbors, d_neighborList,
                     cutsq, lj1, lj2, nAtom);
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    // Copy back forces
    hipEvent_t outputTransfer_start, outputTransfer_stop;
    hipEventCreate(&outputTransfer_start);
    hipEventCreate(&outputTransfer_stop);

    hipEventRecord(outputTransfer_start, 0);
    CUDA_SAFE_CALL(hipMemcpy(force, d_force, nAtom*sizeof(forceVecType),
            hipMemcpyDeviceToHost));
    hipEventRecord(outputTransfer_stop, 0);
    CUDA_SAFE_CALL(hipEventSynchronize(outputTransfer_stop));

    // Get elapsed time
    float outputTransfer_time = 0.0f;
    hipEventElapsedTime(&outputTransfer_time, outputTransfer_start,
            outputTransfer_stop);
    outputTransfer_time *= 1.e-3;

    // If results are incorrect, skip the performance tests
    cout << "Performing Correctness Check (can take several minutes)\n";
    if (!checkResults<T, forceVecType, posVecType>
            (force, position, neighborList, nAtom))
    {
        return;
    }

    // Begin performance tests
    hipEvent_t kernel_start, kernel_stop;
    hipEventCreate(&kernel_start);
    hipEventCreate(&kernel_stop);
    int passes = op.getOptionInt("passes");
    int iter   = op.getOptionInt("iterations");
    for (int i = 0; i < passes; i++)
    {
        // Other kernels will be involved in true parallel versions
        hipEventRecord(kernel_start, 0);
        for (int j = 0; j < iter; j++)
        {
            compute_lj_force<T, forceVecType, posVecType, useTexture, texReader>
                <<<gridSize, blockSize>>>
                (d_force, d_position, maxNeighbors, d_neighborList, cutsq,
                 lj1, lj2, nAtom);
        }
        hipEventRecord(kernel_stop, 0);
        CUDA_SAFE_CALL(hipEventSynchronize(kernel_stop));

        // get elapsed time
        float kernel_time = 0.0f;
        hipEventElapsedTime(&kernel_time, kernel_start, kernel_stop);
        kernel_time /= (float)iter;
        kernel_time *= 1.e-3; // Convert to seconds

        // Total number of flops
        // Every pair of atoms compute distance - 8 flops
        // totalPairs with distance < cutsq perform an additional 13
        // for force calculation
        double gflops = ((8 * nAtom * maxNeighbors) + (totalPairs * 13)) * 1e-9;

        char atts[64];
        sprintf(atts, "%d_atoms", nAtom);;
        resultDB.AddResult(testName, atts, "GFLOPS", gflops / kernel_time);
        resultDB.AddResult(testName+"_PCIe", atts, "GFLOPS",
                gflops / (kernel_time+inputTransfer_time+outputTransfer_time));

        int numPairs = nAtom * maxNeighbors;
        long int nbytes = (3 * sizeof(T) * (1+numPairs)) + // position data
                          (3 * sizeof(T) * nAtom) + // force for each atom
                          (sizeof(int) * numPairs); // neighbor list
        double gbytes = (double)nbytes / (1000. * 1000. * 1000.);
        resultDB.AddResult(testName + "-Bandwidth", atts, "GB/s", gbytes /
                kernel_time);
        resultDB.AddResult(testName + "-Bandwidth_PCIe", atts, "GB/s",
                gbytes / (kernel_time+inputTransfer_time+outputTransfer_time));

        resultDB.AddResult(testName+"_Parity", atts, "N",
                (inputTransfer_time+outputTransfer_time) / kernel_time);
    }

    // Clean up
    // Host
    CUDA_SAFE_CALL(hipHostFree(position));
    CUDA_SAFE_CALL(hipHostFree(force));
    CUDA_SAFE_CALL(hipHostFree(neighborList));
    // Device
    CUDA_SAFE_CALL(hipUnbindTexture(posTexture));
    CUDA_SAFE_CALL(hipFree(d_position));
    CUDA_SAFE_CALL(hipFree(d_force));
    CUDA_SAFE_CALL(hipFree(d_neighborList));
    CUDA_SAFE_CALL(hipEventDestroy(inputTransfer_start));
    CUDA_SAFE_CALL(hipEventDestroy(inputTransfer_stop));
    CUDA_SAFE_CALL(hipEventDestroy(outputTransfer_start));
    CUDA_SAFE_CALL(hipEventDestroy(outputTransfer_stop));
    CUDA_SAFE_CALL(hipEventDestroy(kernel_start));
    CUDA_SAFE_CALL(hipEventDestroy(kernel_stop));
}
// ********************************************************
// Function: distance
//
// Purpose:
//   Calculates distance squared between two atoms
//
// Arguments:
//   position: atom position information
//   i, j: indexes of the two atoms
//
// Returns:  the computed distance
//
// Programmer: Kyle Spafford
// Creation: July 26, 2010
//
// Modifications:
//
// ********************************************************
template <class T, class posVecType>
inline T distance(const posVecType* position, const int i, const int j)
{
    posVecType ipos = position[i];
    posVecType jpos = position[j];
    T delx = ipos.x - jpos.x;
    T dely = ipos.y - jpos.y;
    T delz = ipos.z - jpos.z;
    T r2inv = delx * delx + dely * dely + delz * delz;
    return r2inv;
}

// ********************************************************
// Function: insertInOrder
//
// Purpose:
//   Adds atom j to current neighbor list and distance list
//   if it's distance is low enough.
//
// Arguments:
//   currDist: distance between current atom and each of its neighbors in the
//             current list, sorted in ascending order
//   currList: neighbor list for current atom, sorted by distance in asc. order
//   j:        atom to insert into neighbor list
//   distIJ:   distance between current atom and atom J
//   maxNeighbors: max length of neighbor list
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: July 26, 2010
//
// Modifications:
//
// ********************************************************
template <class T>
inline void insertInOrder(list<T>& currDist, list<int>& currList,
        const int j, const T distIJ, const int maxNeighbors)
{

    typename list<T>::iterator   it;
    typename list<int>::iterator it2;

    it2 = currList.begin();

    T currMax = currDist.back();

    if (distIJ > currMax) return;

    for (it=currDist.begin(); it!=currDist.end(); it++)
    {
        if (distIJ < (*it))
        {
            // Insert into appropriate place in list
            currDist.insert(it,distIJ);
            currList.insert(it2, j);

            // Trim end of list
            currList.resize(maxNeighbors);
            currDist.resize(maxNeighbors);
            return;
        }
        it2++;
    }
}
// ********************************************************
// Function: buildNeighborList
//
// Purpose:
//   Builds the neighbor list structure for all atoms for GPU coalesced reads
//   and counts the number of pairs within the cutoff distance, so
//   the benchmark gets an accurate FLOPS count
//
// Arguments:
//   nAtom:    total number of atoms
//   position: pointer to the atom's position information
//   neighborList: pointer to neighbor list data structure
//
// Returns:  number of pairs of atoms within cutoff distance
//
// Programmer: Kyle Spafford
// Creation: July 26, 2010
//
// Modifications:
//   Jeremy Meredith, Tue Oct  9 17:35:16 EDT 2012
//   On some slow systems and without optimization, this
//   could take a while.  Give users a rough completion
//   percentage so they don't give up.
//
// ********************************************************
template <class T, class posVecType>
inline int buildNeighborList(const int nAtom, const posVecType* position,
        int* neighborList)
{
    int totalPairs = 0;
    // Build Neighbor List
    // Find the nearest N atoms to each other atom, where N = maxNeighbors
    for (int i = 0; i < nAtom; i++)
    {
        // Print progress every 10% completion.
        if (int((i+1)/(nAtom/10)) > int(i/(nAtom/10)))
            cout << "  " << 10*int((i+1)/(nAtom/10)) << "% done\n";

        // Current neighbor list for atom i, initialized to -1
        list<int>   currList(maxNeighbors, -1);
        // Distance to those neighbors.  We're populating this with the
        // closest neighbors, so initialize to FLT_MAX
        list<T> currDist(maxNeighbors, FLT_MAX);

        for (int j = 0; j < nAtom; j++)
        {
            if (i == j) continue; // An atom cannot be its own neighbor

            // Calculate distance and insert in order into the current lists
            T distIJ = distance<T, posVecType>(position, i, j);
            insertInOrder<T>(currDist, currList, j, distIJ, maxNeighbors);
        }
        // We should now have the closest maxNeighbors neighbors and their
        // distances to atom i. Populate the neighbor list data structure
        // for GPU coalesced reads.
        // The populate method returns how many of the maxNeighbors closest
        // neighbors are within the cutoff distance.  This will be used to
        // calculate GFLOPS later.
        totalPairs += populateNeighborList<T>(currDist, currList, i, nAtom,
                neighborList);
    }
    return totalPairs;
}


// ********************************************************
// Function: populateNeighborList
//
// Purpose:
//   Populates the neighbor list structure for a *single* atom for
//   GPU coalesced reads and counts the number of pairs within the cutoff
//   distance, (for current atom) so the benchmark gets an accurate FLOPS count
//
// Arguments:
//   currDist: distance between current atom and each of its maxNeighbors
//             neighbors
//   currList: current list of neighbors
//   i:        current atom
//   nAtom:    total number of atoms
//   neighborList: pointer to neighbor list data structure
//
// Returns:  number of pairs of atoms within cutoff distance
//
// Programmer: Kyle Spafford
// Creation: July 26, 2010
//
// Modifications:
//
// ********************************************************
template <class T>
inline int populateNeighborList(list<T>& currDist,
        list<int>& currList, const int i, const int nAtom,
        int* neighborList)
{
    int idx = 0;
    int validPairs = 0; // Pairs of atoms closer together than the cutoff

    // Iterate across distance and neighbor list
    typename list<T>::iterator distanceIter = currDist.begin();
    for (list<int>::iterator neighborIter = currList.begin();
            neighborIter != currList.end(); neighborIter++)
    {
        // Populate packed neighbor list
        neighborList[(idx * nAtom) + i] = *neighborIter;

        // If the distance is less than cutoff, increment valid counter
        if (*distanceIter < cutsq)
            validPairs++;

        // Increment idx and distance iterator
        idx++;
        distanceIter++;
    }
    return validPairs;
}
