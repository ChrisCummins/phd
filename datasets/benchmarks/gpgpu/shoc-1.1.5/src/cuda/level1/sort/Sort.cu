#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/types.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "cudacommon.h"
#include <cassert>
#include <iostream>
#include <vector>
#include "Sort.h"
#include "sort_kernel.h"
#include "OptionParser.h"
#include "ResultDatabase.h"

using namespace std;

// ****************************************************************************
// Function: addBenchmarkSpecOptions
//
// Purpose:
//   Add benchmark specific options parsing
//
// Arguments:
//   op: the options parser / parameter database
//
// Returns:  nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void addBenchmarkSpecOptions(OptionParser &op)
{
    ;
}

// ****************************************************************************
// Function: RunBenchmark
//
// Purpose:
//   Executes the radix sort benchmark
//
// Arguments:
//   resultDB: results from the benchmark are stored in this db
//   op: the options parser / parameter database
//
// Returns:  nothing, results are stored in resultDB
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void RunBenchmark(ResultDatabase &resultDB, OptionParser &op)
{

    //Number of key-value pairs to sort, must be a multiple of 1024
    int probSizes[4] = { 1, 8, 48, 96 };

    int size = probSizes[op.getOptionInt("size")-1];
    // Convert to MB
    size = (size * 1024 * 1024) / sizeof(uint);

    // Size of the keys & vals buffers in bytes
    uint bytes = size * sizeof(uint);

    // create input data on CPU
    uint *hKeys;
    uint *hVals;
    hipHostMalloc((void**)&hKeys, bytes);
    hipHostMalloc((void**)&hVals, bytes);

    // Allocate space for block sums in the scan kernel.
    uint numLevelsAllocated = 0;
    uint maxNumScanElements = size;
    uint numScanElts = maxNumScanElements;
    uint level = 0;

    do
    {
        uint numBlocks = max(1, (int) ceil((float) numScanElts / (4
                * SCAN_BLOCK_SIZE)));
        if (numBlocks > 1)
        {
            level++;
        }
        numScanElts = numBlocks;
    }
    while (numScanElts > 1);

    uint** scanBlockSums = (uint**) malloc((level + 1) * sizeof(uint*));
    assert(scanBlockSums != NULL);
    numLevelsAllocated = level + 1;
    numScanElts = maxNumScanElements;
    level = 0;

    do
    {
        uint numBlocks = max(1, (int) ceil((float) numScanElts / (4
                * SCAN_BLOCK_SIZE)));
        if (numBlocks > 1)
        {
            // Malloc device mem for block sums
            CUDA_SAFE_CALL(hipMalloc((void**)&(scanBlockSums[level]),
                    numBlocks*sizeof(uint)));
            level++;
        }
        numScanElts = numBlocks;
    }
    while (numScanElts > 1);

    CUDA_SAFE_CALL(hipMalloc((void**)&(scanBlockSums[level]),
            sizeof(uint)));

    // Allocate device mem for sorting kernels
    uint* dKeys, *dVals, *dTempKeys, *dTempVals;

    CUDA_SAFE_CALL(hipMalloc((void**)&dKeys, bytes));
    CUDA_SAFE_CALL(hipMalloc((void**)&dVals, bytes));
    CUDA_SAFE_CALL(hipMalloc((void**)&dTempKeys, bytes));
    CUDA_SAFE_CALL(hipMalloc((void**)&dTempVals, bytes));

    // Each thread in the sort kernel handles 4 elements
    size_t numSortGroups = size / (4 * SORT_BLOCK_SIZE);

    uint* dCounters, *dCounterSums, *dBlockOffsets;
    CUDA_SAFE_CALL(hipMalloc((void**)&dCounters, WARP_SIZE
            * numSortGroups * sizeof(uint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dCounterSums, WARP_SIZE
            * numSortGroups * sizeof(uint)));
    CUDA_SAFE_CALL(hipMalloc((void**)&dBlockOffsets, WARP_SIZE
            * numSortGroups * sizeof(uint)));

    int iterations = op.getOptionInt("passes");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int it = 0; it < iterations; it++)
    {
        // Initialize host memory to some pattern
        for (uint i = 0; i < size; i++)
        {
            hKeys[i] = hVals[i] = i % 1024;
        }

        // Copy inputs to GPU
        double transferTime = 0.;
        hipEventRecord(start, 0);
        CUDA_SAFE_CALL(hipMemcpy(dKeys, hKeys, bytes, hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(dVals, hVals, bytes, hipMemcpyHostToDevice));
        hipEventRecord(stop, 0);
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        float elapsedTime;
        hipEventElapsedTime(&elapsedTime, start, stop);
        transferTime += elapsedTime * 1.e-3; // convert to seconds

        hipEventRecord(start, 0);
        // Perform Radix Sort (4 bits at a time)
        for (int i = 0; i < SORT_BITS; i += 4)
        {
            radixSortStep(4, i, (uint4*)dKeys, (uint4*)dVals,
                    (uint4*)dTempKeys, (uint4*)dTempVals, dCounters,
                    dCounterSums, dBlockOffsets, scanBlockSums, size);
        }
        hipEventRecord(stop, 0);
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        hipEventElapsedTime(&elapsedTime, start, stop);
        double kernelTime = elapsedTime * 1.e-3;

        // Readback data from device
        hipEventRecord(start, 0);
        CUDA_SAFE_CALL(hipMemcpy(hKeys, dKeys, bytes, hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipMemcpy(hVals, dVals, bytes, hipMemcpyDeviceToHost));
        hipEventRecord(stop, 0);
        CUDA_SAFE_CALL(hipEventSynchronize(stop));
        hipEventElapsedTime(&elapsedTime, start, stop);
        transferTime += elapsedTime * 1.e-3;

        // Test to make sure data was sorted properly, if not, return
        if (! verifySort(hKeys, hVals, size))
        {
            return;
        }

        char atts[1024];
        sprintf(atts, "%ditems", size);
        double gb = (bytes * 2.) / (1000. * 1000. * 1000.);
        resultDB.AddResult("Sort-Rate", atts, "GB/s", gb / kernelTime);
        resultDB.AddResult("Sort-Rate_PCIe", atts, "GB/s",
                gb / (kernelTime + transferTime));
        resultDB.AddResult("Sort-Rate_Parity", atts, "N",
                transferTime / kernelTime);
    }
    // Clean up
    for (int i = 0; i < numLevelsAllocated; i++)
    {
        CUDA_SAFE_CALL(hipFree(scanBlockSums[i]));
    }
    CUDA_SAFE_CALL(hipFree(dKeys));
    CUDA_SAFE_CALL(hipFree(dVals));
    CUDA_SAFE_CALL(hipFree(dTempKeys));
    CUDA_SAFE_CALL(hipFree(dTempVals));
    CUDA_SAFE_CALL(hipFree(dCounters));
    CUDA_SAFE_CALL(hipFree(dCounterSums));
    CUDA_SAFE_CALL(hipFree(dBlockOffsets));
    CUDA_SAFE_CALL(hipEventDestroy(start));
    CUDA_SAFE_CALL(hipEventDestroy(stop));

    free(scanBlockSums);
    CUDA_SAFE_CALL(hipHostFree(hKeys));
    CUDA_SAFE_CALL(hipHostFree(hVals));
}

// ****************************************************************************
// Function: radixSortStep
//
// Purpose:
//   This function performs a radix sort, using bits startbit to
//   (startbit + nbits).  It is designed to sort by 4 bits at a time.
//   It also reorders the data in the values array based on the sort.
//
// Arguments:
//      nbits: the number of key bits to use
//      startbit: the bit to start on, 0 = lsb
//      keys: the input array of keys
//      values: the input array of values
//      tempKeys: temporary storage, same size as keys
//      tempValues: temporary storage, same size as values
//      counters: storage for the index counters, used in sort
//      countersSum: storage for the sum of the counters
//      blockOffsets: storage used in sort
//      scanBlockSums: input to Scan, see below
//      numElements: the number of elements to sort
//
// Returns: nothing
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
void radixSortStep(uint nbits, uint startbit, uint4* keys, uint4* values,
        uint4* tempKeys, uint4* tempValues, uint* counters,
        uint* countersSum, uint* blockOffsets, uint** scanBlockSums,
        uint numElements)
{
    // Threads handle either 4 or two elements each
    const size_t radixGlobalWorkSize   = numElements / 4;
    const size_t findGlobalWorkSize    = numElements / 2;
    const size_t reorderGlobalWorkSize = numElements / 2;

    // Radix kernel uses block size of 128, others use 256 (same as scan)
    const size_t radixBlocks   = radixGlobalWorkSize   / SORT_BLOCK_SIZE;
    const size_t findBlocks    = findGlobalWorkSize    / SCAN_BLOCK_SIZE;
    const size_t reorderBlocks = reorderGlobalWorkSize / SCAN_BLOCK_SIZE;

    radixSortBlocks
        <<<radixBlocks, SORT_BLOCK_SIZE, 4 * sizeof(uint)*SORT_BLOCK_SIZE>>>
        (nbits, startbit, tempKeys, tempValues, keys, values);

    findRadixOffsets
        <<<findBlocks, SCAN_BLOCK_SIZE, 2 * SCAN_BLOCK_SIZE*sizeof(uint)>>>
        ((uint2*)tempKeys, counters, blockOffsets, startbit, numElements,
         findBlocks);

    scanArrayRecursive(countersSum, counters, 16*reorderBlocks, 0,
            scanBlockSums);

    reorderData<<<reorderBlocks, SCAN_BLOCK_SIZE>>>
        (startbit, (uint*)keys, (uint*)values, (uint2*)tempKeys,
        (uint2*)tempValues, blockOffsets, countersSum, counters,
        reorderBlocks);
}

void scanArrayRecursive(uint* outArray, uint* inArray, int numElements,
        int level, uint** blockSums)
{
    // Kernels handle 8 elems per thread
    unsigned int numBlocks = max(1,
            (unsigned int)ceil((float)numElements/(4.f*SCAN_BLOCK_SIZE)));
    unsigned int sharedEltsPerBlock = SCAN_BLOCK_SIZE * 2;
    unsigned int sharedMemSize = sizeof(uint) * sharedEltsPerBlock;

    bool fullBlock = (numElements == numBlocks * 4 * SCAN_BLOCK_SIZE);

    dim3 grid(numBlocks, 1, 1);
    dim3 threads(SCAN_BLOCK_SIZE, 1, 1);

    // execute the scan
    if (numBlocks > 1)
    {
        scan<<<grid, threads, sharedMemSize>>>
           (outArray, inArray, blockSums[level], numElements, fullBlock, true);
    } else
    {
        scan<<<grid, threads, sharedMemSize>>>
           (outArray, inArray, blockSums[level], numElements, fullBlock, false);
    }
    if (numBlocks > 1)
    {
        scanArrayRecursive(blockSums[level], blockSums[level],
                numBlocks, level + 1, blockSums);
        vectorAddUniform4<<< grid, threads >>>
                (outArray, blockSums[level], numElements);
    }
}

// ****************************************************************************
// Function: verifySort
//
// Purpose:
//   Simple cpu routine to verify device results
//
// Arguments:
//
//
// Returns:  nothing, prints relevant info to stdout
//
// Programmer: Kyle Spafford
// Creation: August 13, 2009
//
// Modifications:
//
// ****************************************************************************
bool verifySort(uint *keys, uint* vals, const size_t size)
{
    bool passed = true;

    for (unsigned int i = 0; i < size - 1; i++)
    {
        if (keys[i] > keys[i + 1])
        {
            passed = false;
#ifdef VERBOSE_OUTPUT
            cout << "Failure: at idx: " << i << endl;
            cout << "Key: " << keys[i] << " Val: " << vals[i] << endl;
            cout << "Idx: " << i + 1 << " Key: " << keys[i + 1] << " Val: "
                    << vals[i + 1] << endl;
#endif
        }
    }
    cout << "Test ";
    if (passed)
        cout << "Passed" << endl;
    else
        cout << "Failed" << endl;
    return passed;
}
