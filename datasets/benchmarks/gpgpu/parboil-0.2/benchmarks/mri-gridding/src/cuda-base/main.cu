#include "hip/hip_runtime.h"
/***************************************************************************
 *
 *            (C) Copyright 2010 The Board of Trustees of the
 *                        University of Illinois
 *                         All Rights Reserved
 *
 ***************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "parboil.h"

#include "UDTypes.h"
#include "CUDA_interface.h"
#include "CPU_kernels.h"

#define PI 3.14159265
#define CUERR \
  do { \
    hipError_t err; \
    if ((err = hipGetLastError()) != hipSuccess) { \
      printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__); \
      return 0; \
    } \
  } while (0)

/************************************************************ 
 * This function reads the parameters from the file provided
 * as a comman line argument.
 ************************************************************/
void setParameters(FILE* file, parameters* p){
  fscanf(file,"aquisition.numsamples=%d\n",&(p->numSamples));
  fscanf(file,"aquisition.kmax=%f %f %f\n",&(p->kMax[0]), &(p->kMax[1]), &(p->kMax[2]));
  fscanf(file,"aquisition.matrixSize=%d %d %d\n", &(p->aquisitionMatrixSize[0]), &(p->aquisitionMatrixSize[1]), &(p->aquisitionMatrixSize[2]));
  fscanf(file,"reconstruction.matrixSize=%d %d %d\n", &(p->reconstructionMatrixSize[0]), &(p->reconstructionMatrixSize[1]), &(p->reconstructionMatrixSize[2]));
  fscanf(file,"gridding.matrixSize=%d %d %d\n", &(p->gridSize[0]), &(p->gridSize[1]), &(p->gridSize[2]));
  fscanf(file,"gridding.oversampling=%f\n", &(p->oversample));
  fscanf(file,"kernel.width=%f\n", &(p->kernelWidth));
  fscanf(file,"kernel.useLUT=%d\n", &(p->useLUT));

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  printf("  Total amount of GPU memory: %llu bytes\n", (unsigned long long) deviceProp.totalGlobalMem);
  printf("  Number of samples = %d\n", p->numSamples);
  if (p->numSamples > 10000000 && deviceProp.totalGlobalMem/1024/1024 < 3000) {
    printf("  Need at least 3GB of GPU memory for large dataset\n");
    exit(1);
  }
  printf("  Grid Size = %dx%dx%d\n", p->gridSize[0], p->gridSize[1], p->gridSize[2]);
  printf("  Input Matrix Size = %dx%dx%d\n", p->aquisitionMatrixSize[0], p->aquisitionMatrixSize[1], p->aquisitionMatrixSize[2]);
  printf("  Recon Matrix Size = %dx%dx%d\n", p->reconstructionMatrixSize[0], p->reconstructionMatrixSize[1], p->reconstructionMatrixSize[2]);
  printf("  Kernel Width = %f\n", p->kernelWidth);
  printf("  KMax = %.2f %.2f %.2f\n", p->kMax[0], p->kMax[1], p->kMax[2]);
  printf("  Oversampling = %f\n", p->oversample);
  printf("  GPU Binsize = %d\n", p->binsize);
  printf("  Use LUT = %s\n", (p->useLUT)?"Yes":"No");
}

/************************************************************ 
 * This function reads the sample point data from the kspace
 * and klocation files (and sdc file if provided) into the
 * sample array.
 * Returns the number of samples read successfully.
 ************************************************************/
unsigned int readSampleData(parameters params, FILE* uksdata_f, ReconstructionSample* samples){
  unsigned int i;

  for(i=0; i<params.numSamples; i++){
    if (feof(uksdata_f)){
      break;
    }
    fread((void*) &(samples[i]), sizeof(ReconstructionSample), 1, uksdata_f);
  }

  float kScale[3];
  kScale[0] = float(params.aquisitionMatrixSize[0])/(float(params.reconstructionMatrixSize[0])*float(params.kMax[0]));
  kScale[1] = float(params.aquisitionMatrixSize[1])/(float(params.reconstructionMatrixSize[1])*float(params.kMax[1]));
  kScale[2] = float(params.aquisitionMatrixSize[2])/(float(params.reconstructionMatrixSize[2])*float(params.kMax[2]));

  int size_x = params.gridSize[0];
  int size_y = params.gridSize[1];
  int size_z = params.gridSize[2];

  float ax = (kScale[0]*(size_x-1))/2.0;
  float bx = (float)(size_x-1)/2.0;

  float ay = (kScale[1]*(size_y-1))/2.0;
  float by = (float)(size_y-1)/2.0;

  float az = (kScale[2]*(size_z-1))/2.0;
  float bz = (float)(size_z-1)/2.0;

  for(int n=0; n<i; n++){
    samples[n].kX = floor((samples[n].kX*ax)+bx);
    samples[n].kY = floor((samples[n].kY*ay)+by);
    samples[n].kZ = floor((samples[n].kZ*az)+bz);
  }

  return i;
}


int main (int argc, char* argv[]){
  struct pb_Parameters* prms;
  struct pb_TimerSet timers;

  prms = pb_ReadParameters(&argc,argv);
  pb_InitializeTimerSet(&timers);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  char uksdata[250];
  parameters params;

  FILE* uksfile_f = NULL;
  FILE* uksdata_f = NULL;

  strcpy(uksdata,prms->inpFiles[0]);
  strcat(uksdata,".data");

  uksfile_f = fopen(prms->inpFiles[0],"r");
  if (uksfile_f == NULL){
    printf("ERROR: Could not open %s\n",prms->inpFiles[0]);
    exit(1);
  }

  printf("\nReading parameters\n");

  if (argc >= 2){
    params.binsize = atoi(argv[1]);
  } else { //default binsize value;
    params.binsize = 128;
  }

  setParameters(uksfile_f, &params);

  pb_SwitchToTimer(&timers, pb_TimerID_IO);

  ReconstructionSample* samples; //Input Data
  float* LUT; //use look-up table for faster execution on CPU (intermediate data)
  unsigned int sizeLUT; //set in the function calculateLUT (intermediate data)

  cmplx* gridData; //Output Data
  float* sampleDensity; //Output Data

  cmplx* gridData_gold; //Gold Output Data
  float* sampleDensity_gold; //Gold Output Data

  hipHostMalloc((void**)&samples, params.numSamples*sizeof(ReconstructionSample));
  CUERR;
  if (samples == NULL){
    printf("ERROR: Unable to allocate memory for input data\n");
    exit(1);
  }

  uksdata_f = fopen(uksdata,"rb");

  if(uksdata_f == NULL){
    printf("ERROR: Could not open data file\n");
    exit(1);
  }

  printf("Reading input data from files\n");

  unsigned int n = readSampleData(params, uksdata_f, samples);
  fclose(uksdata_f);

  if (params.useLUT){
    printf("Generating Look-Up Table\n");
    float beta = PI * sqrt(4*params.kernelWidth*params.kernelWidth/(params.oversample*params.oversample) * (params.oversample-.5)*(params.oversample-.5)-.8);
    calculateLUT(beta, params.kernelWidth, &LUT, &sizeLUT);
  }

  int gridNumElems = params.gridSize[0] * params.gridSize[1] * params.gridSize[2];

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  gridData_gold = (cmplx*) calloc (gridNumElems, sizeof(cmplx));
  sampleDensity_gold = (float*) calloc (gridNumElems, sizeof(float));
  if (sampleDensity_gold == NULL || gridData_gold == NULL){
    printf("ERROR: Unable to allocate memory for output data\n");
    exit(1);
  }

  printf("Running gold version\n");

  gridding_Gold(n, params, samples, LUT, sizeLUT, gridData_gold, sampleDensity_gold);

  hipHostMalloc((void**)&gridData, gridNumElems*sizeof(cmplx));
  hipHostMalloc((void**)&sampleDensity, gridNumElems*sizeof(float));
  CUERR;
  if (sampleDensity == NULL || gridData == NULL){
    printf("ERROR: Unable to allocate memory for output data\n");
    exit(1);
  }

  printf("Running CUDA version\n");

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  //Interface function to GPU implementation of gridding
  CUDA_interface(&timers, n, params, samples, LUT, sizeLUT, gridData, sampleDensity);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  int passed=1;
  for (int i=0; i<gridNumElems; i++){
    if(sampleDensity[i] != sampleDensity_gold[i]) {
      passed=0;
      break;
    }
  }
  //(passed) ? printf("Comparing GPU and Gold results... PASSED\n"):printf("Comparing GPU and Gold results... FAILED\n");

  pb_SwitchToTimer(&timers, pb_TimerID_IO);

  FILE* outfile;
  if(!(outfile=fopen(prms->outFile,"w")))
  {
        printf("Cannot open output file!\n");
  } else {
        fwrite(&passed,sizeof(int),1,outfile);
        fclose(outfile);
  }

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);

  if (params.useLUT){
    free(LUT);
  }
  hipHostFree(samples);
  hipHostFree(gridData);
  hipHostFree(sampleDensity);
  free(gridData_gold);
  free(sampleDensity_gold);

  printf("\n");
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(prms);

  return 0;
}
