
#include <parboil.h>
#include <stdio.h>
#include <stdlib.h>

#include "file.h"
#include "gpu_info.h"
#include "spmv_jds.h"
#include "jds_kernels.cu"
#include "convert_dataset.h"

static int generate_vector(float *x_vector, int dim) 
{	
	srand(54321);	
	for(int i=0;i<dim;i++)
	{
		x_vector[i] = (rand() / (float) RAND_MAX);
	}
	return 0;
}

int main(int argc, char** argv) {
	struct pb_TimerSet timers;
	struct pb_Parameters *parameters;
	
	
	
	
	
	printf("CUDA accelerated sparse matrix vector multiplication****\n");
	printf("Original version by Li-Wen Chang <lchang20@illinois.edu> and Shengzhao Wu<wu14@illinois.edu>\n");
	printf("This version maintained by Chris Rodrigues  ***********\n");
	parameters = pb_ReadParameters(&argc, argv);
	if ((parameters->inpFiles[0] == NULL) || (parameters->inpFiles[1] == NULL))
    {
      fprintf(stderr, "Expecting one input filename\n");
      exit(-1);
    }

	
	pb_InitializeTimerSet(&timers);
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	//parameters declaration
	int len;
	int depth;
	int dim;
	int pad=32;
	int nzcnt_len;
	
	//host memory allocation
	//matrix
	float *h_data;
	int *h_indices;
	int *h_ptr;
	int *h_perm;
	int *h_nzcnt;
	//vector
	float *h_Ax_vector;
    float *h_x_vector;
	
	//device memory allocation
	//matrix
	float *d_data;
	int *d_indices;
	int *d_ptr;
	int *d_perm;
	int *d_nzcnt;
	//vector
	float *d_Ax_vector;
    float *d_x_vector;
	
    //load matrix from files
	pb_SwitchToTimer(&timers, pb_TimerID_IO);
	//inputData(parameters->inpFiles[0], &len, &depth, &dim,&nzcnt_len,&pad,
	//    &h_data, &h_indices, &h_ptr,
	//    &h_perm, &h_nzcnt);
	int col_count;
	coo_to_jds(
		parameters->inpFiles[0], // bcsstk32.mtx, fidapm05.mtx, jgl009.mtx
		1, // row padding
		pad, // warp size
		1, // pack size
		1, // is mirrored?
		0, // binary matrix
		1, // debug level [0:2]
		&h_data, &h_ptr, &h_nzcnt, &h_indices, &h_perm,
		&col_count, &dim, &len, &nzcnt_len, &depth
	);
	
  h_Ax_vector=(float*)malloc(sizeof(float)*dim);
  h_x_vector=(float*)malloc(sizeof(float)*dim);
  input_vec( parameters->inpFiles[1],h_x_vector,dim);


	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

	
	hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
	
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//memory allocation
	hipMalloc((void **)&d_data, len*sizeof(float));
	hipMalloc((void **)&d_indices, len*sizeof(int));
	hipMalloc((void **)&d_ptr, depth*sizeof(int));
	hipMalloc((void **)&d_perm, dim*sizeof(int));
	hipMalloc((void **)&d_nzcnt, nzcnt_len*sizeof(int));
	hipMalloc((void **)&d_x_vector, dim*sizeof(float));
	hipMalloc((void **)&d_Ax_vector,dim*sizeof(float));
	hipMemset( (void *) d_Ax_vector, 0, dim*sizeof(float));
	
	//memory copy
	hipMemcpy(d_data, h_data, len*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_indices, h_indices, len*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_perm, h_perm, dim*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_x_vector, h_x_vector, dim*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(jds_ptr_int), h_ptr, depth*sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(sh_zcnt_int), h_nzcnt,nzcnt_len*sizeof(int));
	
	hipDeviceSynchronize();
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	unsigned int grid;
	unsigned int block;
    compute_active_thread(&block, &grid,nzcnt_len,pad, deviceProp.major,deviceProp.minor,
					deviceProp.warpSize,deviceProp.multiProcessorCount);

	
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(spmv_jds_texture), hipFuncCachePreferL1);

	//main execution
	pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);
	
	for (int i=0; i<50; i++)
	{
		hipBindTexture(0, tex_x_float, d_x_vector);
		spmv_jds_texture<<<grid, block>>>(d_Ax_vector,
										 d_data,d_indices,d_perm,
										 d_x_vector,d_nzcnt,dim);
		hipUnbindTexture(tex_x_float);	
	}
	
    CUERR // check and clear any existing errors
	
	hipDeviceSynchronize();
	
	pb_SwitchToTimer(&timers, pb_TimerID_COPY);
	//HtoD memory copy
	hipMemcpy(h_Ax_vector, d_Ax_vector,dim*sizeof(float), hipMemcpyDeviceToHost);	

	hipDeviceSynchronize();
	hipFree(d_data);
    hipFree(d_indices);
    hipFree(d_ptr);
	hipFree(d_perm);
    hipFree(d_nzcnt);
    hipFree(d_x_vector);
	hipFree(d_Ax_vector);
 
	if (parameters->outFile) {
		pb_SwitchToTimer(&timers, pb_TimerID_IO);
		outputData(parameters->outFile,h_Ax_vector,dim);
		
	}
	pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
	
	free (h_data);
	free (h_indices);
	free (h_ptr);
	free (h_perm);
	free (h_nzcnt);
	free (h_Ax_vector);
	free (h_x_vector);
	pb_SwitchToTimer(&timers, pb_TimerID_NONE);

	pb_PrintTimerSet(&timers);
	pb_FreeParameters(parameters);

	return 0;

}
